#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

//Device code
__global__
void add(int *a, int *b, int *c) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
c[index] = a[index] + b[index];
}

//Host code
int main(int argc, char* argv[]) {

int i, size, tot_n, tot_size, blocknum, threadnum;
int *a, *b, *c; // host copies of a, b, c
int *d_a, *d_b, *d_c; // device copies of a, b, c

size = sizeof(int);
blocknum = atoi(argv[1]);
threadnum = atoi(argv[2]);
tot_n = blocknum*threadnum;
tot_size = size*tot_n;

a = (int *) malloc(tot_size);
b = (int *) malloc(tot_size);
c = (int *) malloc(tot_size);

for(i=0; i<tot_n; i++){
a[i] = i;
b[i] = tot_n-i;
c[i] = 0;
};


// Allocate space for device copies of a, b, c
hipMalloc(&d_a, tot_size);
hipMalloc(&d_b, tot_size);
hipMalloc(&d_c, tot_size);

// Copy inputs to device
hipMemcpy(d_a, a, tot_size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, tot_size, hipMemcpyHostToDevice);

// Launch add() kernel on GPU
add<<<blocknum,threadnum>>>(d_a, d_b, d_c);

// Copy result back to host
hipMemcpy(c, d_c, tot_size, hipMemcpyDeviceToHost);

// Print results
printf("blocknum = %d\n", blocknum);
printf("threadnum = %d\n", threadnum);
printf("c = \n");
for(i=0; i<tot_n; i++){
printf("%d\n", c[i]);
};

// Cleanup
hipFree(d_a); hipFree(d_b); hipFree(d_c);
free(a); free(b); free(c);
return 0;
}
