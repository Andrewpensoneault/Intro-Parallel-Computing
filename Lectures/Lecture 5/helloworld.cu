#include <stdio.h>
#include <hip/hip_runtime.h>
//Device code
__global__
void cuda_kernel()
{
    printf("Hello, World! \n");
}
//Host code
int main()
{
    cuda_kernel <<< 1, 1 >>> ();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}