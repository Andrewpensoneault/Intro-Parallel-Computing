
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

//Device code
__global__
void add(int *a, int *b, int *c) {
c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

//Host code
int main(int argc, char* argv[]) {

int i, size, tot_n, tot_size;
int *a, *b, *c; // host copies of a, b, c
int *d_a, *d_b, *d_c; // device copies of a, b, c

size = sizeof(int);
tot_n = atoi(argv[1]);
tot_size = size*tot_n;

a = (int *) malloc(tot_size);
b = (int *) malloc(tot_size);
c = (int *) malloc(tot_size);

for(i=0; i<tot_n; i++){
a[i] = i;
b[i] = tot_n-i;
c[i] = 0;
};


// Allocate space for device copies of a, b, c
hipMalloc(&d_a, tot_size);
hipMalloc(&d_b, tot_size);
hipMalloc(&d_c, tot_size);

// Copy inputs to device
hipMemcpy(d_a, a, tot_size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, tot_size, hipMemcpyHostToDevice);

// Launch add() kernel on GPU
add<<<tot_n,1>>>(d_a, d_b, d_c);

// Copy result back to host
hipMemcpy(c, d_c, tot_size, hipMemcpyDeviceToHost);

// Print results
printf("c = \n");
for(i=0; i<tot_n; i++){
printf("%d\n", c[i]);
};

// Cleanup
hipFree(d_a); hipFree(d_b); hipFree(d_c);
free(a); free(b); free(c);
return 0;
}
